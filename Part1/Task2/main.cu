
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
// ядро
__global__ void add( int *a, int *b, int *c ) {
    *c = *a + *b;
}
//главная функция
int main()
{
    // переменные на CPU
    int a, b, c;
    printf("Введите а:\n");
    std::cin >> a;
    printf("Введите b:\n");
    std::cin >> b;
    // переменные на GPU
    int *dev_a, *dev_b, *dev_c;
    int size = sizeof( int );
    //размерность
    // выделяем память на GPU
    hipMalloc(
            (void**)&dev_a, size );
    hipMalloc(
            (void**)&dev_b, size );
    hipMalloc(
            (void**)&dev_c, size );
// инициализация переменных
// копирование информации с CPU на GPU
    hipMemcpy( dev_a, &a, size, hipMemcpyHostToDevice
    );
    hipMemcpy( dev_b, &b, size, hipMemcpyHostToDevice
    );
// вызов ядра
    add<<< 1, 1 >>>( dev_a, dev_b, dev_c );
// копирование результата работы ядра с GPU на CPU
    hipMemcpy( &c, dev_c, size, hipMemcpyDeviceToHost
    );
// вывод информации
    printf("%d + %d = %d\n", a, b, c);
// очищение памяти на GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}
