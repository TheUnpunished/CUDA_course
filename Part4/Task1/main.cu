
#include <hip/hip_runtime.h>
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include "chrono"
#include "math.h"

__device__ float my_rand(hiprandState state){
    return hiprand_uniform(&state);
}

__global__ void kernel_init(hiprandState *state_x, hiprandState *state_y, int offset){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(clock64(), idx, offset * 2, &state_x[idx]);
    hiprand_init(clock64(), idx, offset * 2 + 1, &state_y[idx]);
}

__global__ void methodMC(int *mCarlo, int n, int nThreads, int nBlocks,
                         hiprandState *state_x, hiprandState *state_y, int offset){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx + offset * nThreads * nBlocks <= n){
        float x = my_rand(state_x[idx]);
        float y = my_rand(state_y[idx]);
//        printf("%f %f \n", x, y);
        if(x * x + y * y <= 1){
            atomicAdd(mCarlo, 1);
        }
    }
}

int main() {
    int nThreads = 200;
    int nBlocks = 512;
    dim3 dimBlock(nThreads, 1,1);
    dim3 dimGrid(nBlocks, 1, 1);
    int n = 10000000;
    int * mCarlo_dev;
    hipMalloc((void **) &mCarlo_dev, sizeof(int));
    hipMemset(mCarlo_dev, 0, sizeof(int));
    int offset_max = n / (nBlocks * nThreads);
    if(n % (nBlocks * nThreads) != 0){
        offset_max ++;
    }
    auto start = std::chrono::high_resolution_clock::now();
    for(int offset = 0; offset < offset_max; offset ++){
        hiprandState *state_x;
        hiprandState *state_y;
        hipMalloc((void **) &state_x, sizeof(hiprandState) * nThreads * nBlocks);
        hipMalloc((void **) &state_y, sizeof(hiprandState) * nThreads * nBlocks);
        kernel_init<<<dimGrid, dimBlock>>>(state_x, state_y, offset);
        hipDeviceSynchronize();
        methodMC<<<dimGrid, dimBlock>>>(mCarlo_dev, n, nThreads, nBlocks,
                                        state_x, state_y, offset);
        hipDeviceSynchronize();
        hipFree(state_x);
        hipFree(state_y);
    }
    int mCarlo;
    hipMemcpy(&mCarlo, mCarlo_dev, sizeof(int), hipMemcpyDeviceToHost);
    double pi = (double) mCarlo / (double) n * 4;
    auto stop = std::chrono::high_resolution_clock::now();
    printf("Pi = %.15f\n", pi);
    printf("Diff = %.15f\n", abs(M_PI - pi));
    printf("Time: ");
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
    printf(" ms\n");
    hipFree(mCarlo_dev);
    return 0;
}
