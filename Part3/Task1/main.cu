#include "hip/hip_runtime.h"

#include <stdio.h>
__global__ void HelloWorld()
{
    printf("Hello world, %d, %d\n", blockIdx.x,
           threadIdx.x);
}
int main()
{
    HelloWorld <<<2, 5>>>();
// хост ожидает завершения работы девайса
    hipDeviceSynchronize();
// ожидаем нажатия любой клавиши
    getchar();
    return 0;
}

