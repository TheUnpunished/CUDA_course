
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include "chrono"



int main() {
    unsigned char *data_local = NULL;
    int iterations = 100;
    hipEvent_t start, stop;
    float elapsedTime = 0.0f;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t memSize = deviceProp.totalGlobalMem / 2;
    data_local = (unsigned char *) malloc(memSize);
    for(unsigned int i = 0; i < memSize / sizeof(unsigned char); i++){
        data_local[i] = (unsigned char)(i & 0xff);
    }
    hipEventCreate(&start);
    hipEventCreate(&stop);
    unsigned char *data_device;
    hipMalloc((void **) &data_device, memSize);
    hipEventRecord(start, 0);
    for(int i = 0; i < iterations; i ++){
        hipMemcpy(data_device, data_local, memSize, hipMemcpyHostToDevice);
    }
    hipEventRecord(stop, 0);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Время: %f\n", elapsedTime);
    float bandWidth = ((float)(1024) * memSize * (float)iterations) /
                      (elapsedTime * (float)(1 << 30));
    printf("Проп. способность: %f ГБ/с", bandWidth);
    hipFree(data_device);
    free(data_local);
    return 0;
}
