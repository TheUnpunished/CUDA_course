
#include <hip/hip_runtime.h>
#include <iostream>
#include "math.h"
#include "chrono"

__global__ void zeta(double *sum, int n, double s, int nThreads, int nBlocks){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    double x;
    for(int i = index; i < n; i += nThreads * nBlocks){
        x = 1.0 / pow((double) i + 1, s);
        sum[index] += x;
    }
}


int main() {
    int nThreads = 512;
    int nBlocks = 30000;
    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(nThreads, 1, 1);
    printf("Введите число шагов: ");
    int n = 1000000;
    std::cin >> n;
    printf("Введите s: ");
    double s = 2.0;
    std::cin >> s;
    size_t sum_size = nThreads * nBlocks * sizeof(double);
    double *sum_local = (double*) malloc(sum_size);
    double *sum_dev;
    hipMalloc((void**) &sum_dev, sum_size);
    hipMemset(sum_dev, 0, sum_size);
    auto start = std::chrono::high_resolution_clock::now();
    zeta<<<dimGrid, dimBlock>>>(sum_dev, n, s, nThreads, nBlocks);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("%s ",
                                   hipGetErrorString(err));
    hipMemcpy(sum_local, sum_dev, sum_size, hipMemcpyDeviceToHost);
    double func = 0.0;
    for(int i = 0; i < nThreads * nBlocks; i ++){
        func += sum_local[i];
    }
    auto stop = std::chrono::high_resolution_clock::now();
    printf("Значение дзета функции Римана: %.20f", func);
    printf("Time ");
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
    printf(" ms\n");
    hipFree(sum_dev);
    free(sum_local);
    return 0;
}
