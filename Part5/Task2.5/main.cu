
#include <hip/hip_runtime.h>
#include <iostream>
#include "ctime"
#include "chrono"

__global__ void scalar(double *x, double *y, double *sum, int n, int nThreads, int nBlocks){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    double m;
    for (int i = idx; i < n; i += nThreads * nBlocks){
        m = x[i] * y[i];
        sum[idx] += m;
    }
}

int main() {
    std::srand(time(nullptr));
    int n = 100000;
    int nThreads = 960;
    int nBlocks = 100;
    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(nThreads, 1,1);
    double x[n];
    double y[n];
    for(int i = 0; i < n; i ++){
        x[i] = ((double) (std::rand() % 100000001) / 100000) - 500;
        y[i] = ((double) (std::rand() % 100000001) / 100000) - 500;
    }
    auto start = std::chrono::high_resolution_clock::now();
    size_t array_size = sizeof(double) * n;
    size_t return_size = sizeof(double) * nThreads * nBlocks;
    double *x_dev;
    double *y_dev;
    double *sum_dev;
    hipMalloc((void **) &x_dev, array_size);
    hipMalloc((void **) &y_dev, array_size);
    hipMalloc((void **) &sum_dev, return_size);
    hipMemset(sum_dev, 0.0, return_size);
    hipMemcpy(x_dev, x, array_size, hipMemcpyHostToDevice);
    hipMemcpy(y_dev, y, array_size, hipMemcpyHostToDevice);
    scalar<<<dimGrid, dimBlock>>>(x_dev, y_dev, sum_dev, n, nThreads, nBlocks);
    hipDeviceSynchronize();
    double *sum = (double*) malloc(return_size);
    hipMemcpy(sum, sum_dev, return_size, hipMemcpyDeviceToHost);
    double scalar_total = 0;
    for(int i = 0; i < nThreads * nBlocks; i ++){
        scalar_total += sum[i];
    }
    auto stop = std::chrono::high_resolution_clock::now();
    printf("Scalar sum = %.3f\n", scalar_total);
    printf("Time: ");
    std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
    printf(" ms\n");
    hipFree(sum_dev);
    hipFree(y_dev);
    hipFree(x_dev);
    free(sum);
    return 0;
}
