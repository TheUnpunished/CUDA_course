
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define BLOCK_SIZE 32
#define GRID_SIZE 16
#define BASE_TYPE int
#define n 100000

__global__ void scalar (BASE_TYPE* a, BASE_TYPE* sum, int offset){
    int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    idx += offset * BLOCK_SIZE * GRID_SIZE;
    __shared__ BASE_TYPE a_sh[BLOCK_SIZE];
    if(idx < n)
        a_sh[threadIdx.x] = a[idx];
    else
        a_sh[threadIdx.x] = 0.0;
    __syncthreads();
    if(threadIdx.x == 0){
        BASE_TYPE sum_temp = 0.0;
        for(int i = 0; i < BLOCK_SIZE; i ++)
            sum_temp += a_sh[i] * a_sh[i];
        sum[blockIdx.x] = sum_temp;
    }
}

int main() {
    dim3 dimBlock(BLOCK_SIZE, 1, 1);
    dim3 dimGrid(GRID_SIZE, 1, 1);
    std::srand(time(nullptr));
    size_t size = sizeof(BASE_TYPE) * n;
    size_t size_sum = sizeof(BASE_TYPE) * GRID_SIZE;
    BASE_TYPE* a = (BASE_TYPE*) malloc(size);
    for(int i = 0; i < n; i ++){
        a[i] = std::rand() % 201 - 100;
    }
    BASE_TYPE* a_dev;
    hipMalloc((void **) &a_dev, size);
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    BASE_TYPE* sum_dev;
    hipMalloc((void **) &sum_dev, size_sum);
    int offset_max;
    if((n) % (BLOCK_SIZE * GRID_SIZE) != 0)
        offset_max = n / (BLOCK_SIZE * GRID_SIZE) + 1;
    else
        offset_max = n / (BLOCK_SIZE * GRID_SIZE);
    BASE_TYPE sum = 0.0;
    BASE_TYPE* sum_h = (BASE_TYPE*) malloc(size_sum);
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for(int offset = 0; offset <= offset_max; offset ++){
        scalar<<<dimGrid, dimBlock>>>(a_dev, sum_dev, offset);
        hipMemcpy(sum_h, sum_dev, size_sum, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for(int i = 0; i < GRID_SIZE; i ++){
            sum += sum_h[i];
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    free(a);
    free(sum_h);
    hipFree(a_dev);
    hipFree(sum_dev);
    std::cout << sum << std::endl;
    double sum_sqrt = sqrt(sum);
    std::cout << sum_sqrt << std::endl;
    printf("Время работы на ГПУ: %.2f мс\n", elapsedTime);
    return 0;
}
